#include "hip/hip_runtime.h"
﻿#include "nvrtcRoutines.h"

#include <hip/hip_runtime.h>

#include <iostream>

// --- Loading an ASCII file and assigning it to a string
#include <string>
#include <fstream>
#include <streambuf>

#define BLOCKSIZE 8

/**********/
/* iDivUp */
/**********/
int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/******************/
/* CUDA SAFE CALL */
/******************/
#define gpuErrchk(x) \
do { \
	hipError_t result = x; \
	if (result != hipSuccess) { \
		const char *msg; \
		hipDrvGetErrorName(result, &msg); \
		std::cerr << "\nerror: " #x " failed with error " << msg << '\n'; \
		exit(1); } \
 } while(0)

/**************************************/
/* GLOBAL FUNCTIONS DEFINED BY STRING */
/**************************************/
const char* kernels = " \n\
extern \"C\" __global__ \n\
void kernel1(float *d_x, float *d_y, float *d_c, int N) \n\
{ \n\
 const int tid = blockIdx.x * blockDim.x + threadIdx.x; \n\
 if (tid < N) { \n\
 d_c[tid] = d_x[tid] + d_y[tid]; \n\
 } \n\
} \n\
extern \"C\" __global__ void kernel2(float *d_x, float *d_y, float *d_c, int N) \n\
{ \n\
 const int tid = blockIdx.x * blockDim.x + threadIdx.x; \n\
 if (tid < N) { \n\
 d_c[tid] = 2.f * d_x[tid] + 2.f * d_y[tid]; \n\
 } \n\
} \n";

	

/********/
/* MAIN */
/********/
int main() {

	int N = 21;
	
	// --- Loading an ASCII file containing the relevant kernels
	std::ifstream t("example.txt");
	std::string str((std::istreambuf_iterator<char>(t)), std::istreambuf_iterator<char>());
	std::cout << str << std::endl;

	// --- Compiling to ptx string
	//char* ptx = compile2PTX((char *)str.c_str());
	char* ptx = compile2PTX(kernels);
	//std::cout << ptx << std::endl;

	// --- Host array allocation and initialization 
	float* h_x = (float*)malloc(N * sizeof(float));
	float* h_y = (float*)malloc(N * sizeof(float));
	float* h_z = (float*)malloc(N * sizeof(float));
	float* h_w = (float*)malloc(N * sizeof(float));
	for (int i = 0; i < N; i++) {
		h_x[i] = (float)i;
		h_y[i] = (float)(i * 2);
	}

	// --- Load the generated PTX and get handle to the SAXPY kernel.
	gpuErrchk(hipInit(0));
	hipDevice_t   cuDevice; gpuErrchk(hipDeviceGet(&cuDevice, 0));
	hipCtx_t  context;  gpuErrchk(hipCtxCreate(&context, 0, cuDevice));
	hipModule_t   module;   gpuErrchk(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));
	hipFunction_t kernel1;  gpuErrchk(hipModuleGetFunction(&kernel1, module, "kernel1"));
	hipFunction_t kernel2;  gpuErrchk(hipModuleGetFunction(&kernel2, module, "kernel2"));

	// --- Device array allocation.
	hipDeviceptr_t d_x; gpuErrchk(hipMalloc(&d_x, N * sizeof(float)));
	hipDeviceptr_t d_y; gpuErrchk(hipMalloc(&d_y, N * sizeof(float)));
	hipDeviceptr_t d_z; gpuErrchk(hipMalloc(&d_z, N * sizeof(float)));
	hipDeviceptr_t d_w; gpuErrchk(hipMalloc(&d_w, N * sizeof(float)));

	// --- Host-device mem copies.
	gpuErrchk(hipMemcpyHtoD(d_x, h_x, N * sizeof(float)));
	gpuErrchk(hipMemcpyHtoD(d_y, h_y, N * sizeof(float)));
	
	// --- Execute kernels.
	void* args1[] = { &d_x, &d_y, &d_z, &N };
	void* args2[] = { &d_x, &d_y, &d_w, &N };
	gpuErrchk(hipModuleLaunchKernel(kernel1,
			iDivUp(N, BLOCKSIZE), 1, 1,	// --- grid dim
			BLOCKSIZE,  1, 1,			// --- block dim
			0, NULL,					// --- shared mem and stream
			args1, 0));					// --- arguments
	gpuErrchk(hipModuleLaunchKernel(kernel2,
		iDivUp(N, BLOCKSIZE), 1, 1,	// --- grid dim
		BLOCKSIZE, 1, 1,			// --- block dim
		0, NULL,					// --- shared mem and stream
		args2, 0));					// --- arguments
	gpuErrchk(hipCtxSynchronize());
	
	// --- Device-host mem copies.
	gpuErrchk(hipMemcpyDtoH(h_z, d_z, N * sizeof(float)));
	gpuErrchk(hipMemcpyDtoH(h_w, d_w, N * sizeof(float)));

	for (int i = 0; i < N; i++) {
		std::cout << i << "\t" << h_x[i] << "\t" << h_y[i] << "\t" << h_z[i] << "\t" << h_w[i] << '\n';
	}
	
	// --- Release resources.
	gpuErrchk(hipFree(d_x));
	gpuErrchk(hipFree(d_y));
	gpuErrchk(hipFree(d_z));
	gpuErrchk(hipFree(d_w));
	gpuErrchk(hipModuleUnload(module));
	gpuErrchk(hipCtxDestroy(context));
	free(h_x);
	free(h_y);
	free(h_z);
	free(h_w);

	return 0;
}